#include "hip/hip_runtime.h"
#include "kernel.h"



inline int divUp(int a, int b)
{
    assert(b > 0);
    return ceil((float)a / b);
};

template<typename Dtype>
__global__ void subscal_kernel(const unsigned int sample_cnt, void* data, float scale, float mean)
{
    Dtype* idata = static_cast<Dtype*>(data);
    unsigned int cur_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (cur_id < sample_cnt)
        idata[cur_id] = (idata[cur_id] - mean) * scale;
}


void subscal(const unsigned int sample_cnt, void* data, float scale, float mean, const hipStream_t& stream)
{
    unsigned int n_blocks = (sample_cnt + (CUDA_BLOCK_THREADS - 1)) / CUDA_BLOCK_THREADS;
    subscal_kernel<float><<<n_blocks, CUDA_BLOCK_THREADS, 0, stream>>>(
        sample_cnt, data, scale, mean
    );
}

__device__ __forceinline__ float2 calcCoord(const float *c_warpMat, int x, int y)
{
    const float coeff = 1.0f / (c_warpMat[6] * x + c_warpMat[7] * y + c_warpMat[8]);
    const float xcoo = coeff * (c_warpMat[0] * x + c_warpMat[1] * y + c_warpMat[2]);
    const float ycoo = coeff * (c_warpMat[3] * x + c_warpMat[4] * y + c_warpMat[5]);

    return make_float2(xcoo, ycoo);
}

template<typename T>
__global__ void warp(ImagePtrCUDA<T> srcPtr, ImagePtrCUDA<float> dst, ImagePtrCUDA<float> dst_gray, float** ptMatrixPtr, int* ploy2Imgs, bool upsidedown, bool onlyRGB=false)
{
    
    const int  dst_b = blockIdx.z;
    const int src_b = ploy2Imgs[dst_b];
    const int  x     = blockDim.x * blockIdx.x + threadIdx.x;
    const int  y     = blockDim.y * blockIdx.y + threadIdx.y;
    const int  c     = threadIdx.z;
    if (x < dst.cols && y < dst.rows)
    {
        const float2 coord = calcCoord(ptMatrixPtr[dst_b], x, y);
        *dst.ptr(dst_b, y, x, c) = linearInterp<T, float>(srcPtr, src_b, coord.y, coord.x, c);
        __syncthreads();
        
        int b   = *dst.ptr(dst_b, y, x, 0);
        int g   = *dst.ptr(dst_b, y, x, 1);
        int r   = *dst.ptr(dst_b, y, x, 2);
        if (onlyRGB)
        {
            // BGR to RGB and norm
            // mean value from https://github.com/mlfoundations/open_clip/blob/b4cf9269b0b11c0eea47cb16039369a46bd67449/src/open_clip/constants.py
            *dst.ptr(dst_b, y, x, 0) = (r/255.0 - 0.4814546)/0.26862954;
            *dst.ptr(dst_b, y, x, 1) = (g/255.0 - 0.4578275)/0.26130258;
            *dst.ptr(dst_b, y, x, 2) = (b/255.0 - 0.4082107)/0.27577711;
            return ;
        }
        // bgr to gray
        T gray  = (T)CV_DESCALE(b * BY15 + g * GY15 + r * RY15, GRAY_SHIFT);
        float gray_scale   = ((float)gray - IMG_MEAN_GRAY) * IMG_SCALE_GRAY;
        *dst_gray.ptr(dst_b, y, x, 0) = gray_scale;
        if(upsidedown)
        {
            int dst_b_UD = dst_b + dst.batches;
            int x_UD = (dst.cols - 1 - x);
            int y_UD = (dst.rows - 1 - y);
            *dst_gray.ptr(dst_b_UD, y_UD, x_UD, 0) = gray_scale;
        }
    }
}

template<typename T>
struct WarpDispatcher
{
    static void call(const ImagePtrCUDA<T> src, ImagePtrCUDA<float> dst, ImagePtrCUDA<float> dst_gray, float** ptMatrixPtr, int* ploy2Imgs,  bool upsidedown, bool onlyRGB,
                     const hipStream_t& stream)
    {

        dim3 block(BLOCK, BLOCK / 4, 3);
        dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y), dst.batches);

        // share memory  to save persceptive trans matrix ptr
        size_t smem_size = dst.batches * sizeof(float*);
        warp<<<grid, block, smem_size, stream>>>(src, dst, dst_gray, ptMatrixPtr, ploy2Imgs, upsidedown, onlyRGB);
        hipStreamSynchronize(stream);
        checkKernelErrors();
    }
};

void warp_caller(const ImagePtrCUDA<uchar>& src, ImagePtrCUDA<float>& dst, ImagePtrCUDA<float>& dst_gray, float** ptMatrixPtr, int* poly2Imgs, bool upsidedown, bool isRGBInput, const hipStream_t& stream)
{
    WarpDispatcher<uchar>::call(src, dst, dst_gray, ptMatrixPtr, poly2Imgs, upsidedown, isRGBInput, stream);
}



__global__ void nhwc2nchwWithMeanScaleResize(ImagePtrCUDA<uchar> src , ImagePtrCUDA<float> dst)
{
    const int dst_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int dst_b     = blockIdx.z;
    const int dst_c     = threadIdx.z;
    const float scale_x = ((float)src.cols) / dst.cols;
    const float scale_y = ((float)src.rows) / dst.rows;

    if (dst_x < dst.cols && dst_y < dst.rows)
    {
        float fy = (float)((dst_y + 0.5f) * scale_y - 0.5f);
        float fx = (float)((dst_x + 0.5f) * scale_x - 0.5f);
        *dst.ptr(dst_b, dst_y, dst_x, dst_c) = linearInterp<uchar, float>(src, dst_b, fy, fx, dst_c);
       
        if(dst_c == 0)
        {
            *dst.ptr(dst_b, dst_y, dst_x, dst_c) -= IMG_MEAN_B;
        }
        else if(dst_c == 1)
        {
            *dst.ptr(dst_b, dst_y, dst_x, dst_c) -= IMG_MEAN_G;
        }
        else if (dst_c == 2)
        {
            *dst.ptr(dst_b, dst_y, dst_x, dst_c) -= IMG_MEAN_R;
        }
        *dst.ptr(dst_b, dst_y, dst_x, dst_c) *= IMG_SCALE_BRG;
    }
    return;
}


__global__ void threshold(ImagePtrCUDA<float> src , ImagePtrCUDA<uchar> dst, const float binaryThreshold)
{
    const int dst_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int dst_b     = blockIdx.z;
    const int dst_c     = threadIdx.z;

    if (dst_x < dst.cols && dst_y < dst.rows)
    {
        if(*src.ptr(dst_b, dst_y, dst_x, dst_c) > binaryThreshold)
        {
            *dst.ptr(dst_b, dst_y, dst_x, dst_c) =   255;
        }
        else
        {
            *dst.ptr(dst_b, dst_y, dst_x, dst_c) =   0;
        }
    }

}

void blobFromImagesCUDA(void* inData, void* outData, const nvinfer1::Dims& inShape, const nvinfer1::Dims& outShape, bool inputIsNHWC, const hipStream_t& stream)
{
    int outBatchSize = outShape.d[0];
    int outChannels = outShape.d[1];
    int outHeight = outShape.d[2];
    int outWidth = outShape.d[3];

    int inBatchSize = inShape.d[0];
    int inHeight = 0;
    int inWeight = 0;
    int inChannels = 0;
    if (inputIsNHWC)
    {
        inHeight = inShape.d[1];
        inWeight = inShape.d[2];
        inChannels = inShape.d[3];
    }
    else
    {
        inHeight = inShape.d[2];
        inWeight = inShape.d[3];
        inChannels = inShape.d[1];
    }
    ImagePtrCUDA<uchar> srcPtr(inBatchSize, inHeight, inWeight, inChannels, static_cast<uchar*>(inData), inputIsNHWC);
    ImagePtrCUDA<float> dstPtr(outBatchSize, outHeight, outWidth, outChannels, static_cast<float*>(outData), false);
    dim3 block(BLOCK, BLOCK / 4, 3);
    dim3 grid(divUp(dstPtr.cols, block.x), divUp(dstPtr.rows, block.y), dstPtr.batches);

    nhwc2nchwWithMeanScaleResize<<<grid, block, 0, stream>>>(srcPtr, dstPtr);
    hipStreamSynchronize(stream);
    checkKernelErrors();
}


void thresholdCUDA(void* inData, void* outData, const nvinfer1::Dims& inShape, const float binaryThreshold, const hipStream_t& stream)
{
    int inBatchSize = inShape.d[0];
    int inChannels = inShape.d[1];
    int inHeight = inShape.d[2];
    int inWeight = inShape.d[3];
    ImagePtrCUDA<float> srcPtr(inBatchSize, inHeight, inWeight, inChannels, static_cast<float*>(inData), false);
    ImagePtrCUDA<uchar> dstPtr(inBatchSize, inHeight, inWeight, inChannels, static_cast<uchar*>(outData), false);
    dim3 block(BLOCK, BLOCK / 4, 1);
    dim3 grid(divUp(srcPtr.cols, block.x), divUp(srcPtr.rows, block.y), srcPtr.batches);
    threshold<<<grid, block, 0, stream>>>(srcPtr, dstPtr, binaryThreshold);

    hipStreamSynchronize(stream);
    checkKernelErrors();
}

__global__ void calculateRotateCoef(float *aCoeffs, const int degrees)
{
    int angle = blockIdx.x * blockDim.x + threadIdx.x;
    if(angle < degrees)
    {
        aCoeffs[2 * angle]      = cos(angle * PI / 180);
        aCoeffs[2 * angle + 1]  = sin(angle * PI / 180);
    }
}

void calculateRotateCoefCUDA(void* rotateCoefBuf, const int degrees,  const hipStream_t& stream)
{
    dim3 block(BLOCK * 8);
    dim3 grid(divUp(degrees, block.x));
    float* aCoeffs = static_cast<float*>(rotateCoefBuf);
    calculateRotateCoef<<<grid, block, 0, stream>>>(aCoeffs, degrees);
    
}

__global__ void calculateRotateArea(ContourPtrCUDA<uint16_t> inContourPointsData, MinAreaRectPtrCUDA<int> outMinAreaRectBox, float* rotateCoeffs, int* numPointsInContourBuf, int* contoursToImagesBuf)
{
   
    int pointIdx    = blockIdx.x * blockDim.x + threadIdx.x;
    int contourIdx  = blockIdx.y;
    
    int angleIdx       = blockIdx.z;
    extern  __shared__ float rotateCoeffs_sm[];
    rotateCoeffs_sm[2 * angleIdx] = rotateCoeffs[2 * angleIdx];
    rotateCoeffs_sm[2 * angleIdx + 1] = rotateCoeffs[2 * angleIdx + 1];
    __syncthreads();
    
    if(pointIdx < numPointsInContourBuf[contourIdx])
    {
        float px = *inContourPointsData.ptr(contourIdx, pointIdx, 0);
        float py = *inContourPointsData.ptr(contourIdx, pointIdx, 1);
        float cos_coeff = rotateCoeffs_sm[2 * angleIdx];
        float sin_coeff = rotateCoeffs_sm[2 * angleIdx + 1];
        int px_rot = (px * cos_coeff) - (py * sin_coeff);
        int py_rot = (px * sin_coeff) + (py * cos_coeff);
        //xmin
        atomicMin(outMinAreaRectBox.pointPtr(contourIdx,angleIdx,0),px_rot);
        //ymin
        atomicMin(outMinAreaRectBox.pointPtr(contourIdx,angleIdx,1),py_rot);
        //xmax
        atomicMax(outMinAreaRectBox.pointPtr(contourIdx,angleIdx,2),px_rot);
        //ymax
        atomicMax(outMinAreaRectBox.pointPtr(contourIdx,angleIdx,3),py_rot);
        // TODO(@binz) can this func sync across all global block?
        __threadfence();
        int rectWidth = *outMinAreaRectBox.pointPtr(contourIdx,angleIdx,2) - *outMinAreaRectBox.pointPtr(contourIdx,angleIdx,0);
        int rectHeight = *outMinAreaRectBox.pointPtr(contourIdx,angleIdx,3) - *outMinAreaRectBox.pointPtr(contourIdx,angleIdx,1);
        *outMinAreaRectBox.pointPtr(contourIdx,angleIdx,4) = rectWidth * rectHeight;
        *outMinAreaRectBox.pointPtr(contourIdx,angleIdx,5) = angleIdx;
    }
}

__global__ void findMinAreaAndAngle(MinAreaRectPtrCUDA<int> outMinAreaRectBox, const int numOfDegrees)
{
    int angleIdx = threadIdx.x;
	if (angleIdx > numOfDegrees)
    {
        return;
    }
		
    int rectIdx = blockIdx.x;
    extern __shared__ int areaAngleBuf_sm[];
    areaAngleBuf_sm[2*angleIdx] = *outMinAreaRectBox.pointPtr(rectIdx, angleIdx, 4);
    areaAngleBuf_sm[2*angleIdx + 1] = *outMinAreaRectBox.pointPtr(rectIdx, angleIdx, 5);
    __syncthreads();
   
    for (int stride = numOfDegrees/2; stride >0; stride >>=1)
	{
		if (angleIdx < stride)
		{
            int* curAreaIdx     = &areaAngleBuf_sm[2*angleIdx];
            int* nextAreaIdx    = &areaAngleBuf_sm[2*(angleIdx + stride)];
            int* curAngleIdx    = &areaAngleBuf_sm[2*angleIdx + 1];
            int* nextAngleIdx   = &areaAngleBuf_sm[2*(angleIdx + stride) + 1];
            if(*curAreaIdx > *nextAreaIdx)
            {
                *curAreaIdx = *nextAreaIdx;
                *curAngleIdx = *nextAngleIdx;
            }	    
		}
		__syncthreads();

        if(stride%2 == 1 && areaAngleBuf_sm[0] > areaAngleBuf_sm[2*(stride - 1)])
        {
            areaAngleBuf_sm[0]  = areaAngleBuf_sm[2*(stride - 1) ];
            areaAngleBuf_sm[1]  = areaAngleBuf_sm[2*(stride - 1) + 1];
        }
        __syncthreads();
	}
    if(numOfDegrees%2 == 1 && areaAngleBuf_sm[0] > areaAngleBuf_sm[2*(numOfDegrees - 1)])
    {
        areaAngleBuf_sm[0]  = areaAngleBuf_sm[2*(numOfDegrees - 1) ];
        areaAngleBuf_sm[1]  = areaAngleBuf_sm[2*(numOfDegrees - 1) + 1];
    }

    int minRotateAngle = areaAngleBuf_sm[1];
    float cos_coeff  = cos(- minRotateAngle * PI / 180);
    float sin_coeff  = sin(- minRotateAngle * PI / 180);
    float xmin = *outMinAreaRectBox.pointPtr(rectIdx, areaAngleBuf_sm[1], 0);
    float ymin = *outMinAreaRectBox.pointPtr(rectIdx, areaAngleBuf_sm[1], 1);
    float xmax = *outMinAreaRectBox.pointPtr(rectIdx, areaAngleBuf_sm[1], 2);
    float ymax = *outMinAreaRectBox.pointPtr(rectIdx, areaAngleBuf_sm[1], 3);
    float tl_x = (xmin * cos_coeff) - (ymin * sin_coeff);
    float tl_y = (xmin * sin_coeff) + (ymin * cos_coeff);
    float br_x = (xmax * cos_coeff) - (ymax * sin_coeff);
    float br_y = (xmax * sin_coeff) + (ymax * cos_coeff);
    float tr_x = (xmax * cos_coeff) - (ymin * sin_coeff);
    float tr_y = (xmax * sin_coeff) + (ymin * cos_coeff);
    float bl_x = (xmin * cos_coeff) - (ymax * sin_coeff);
    float bl_y = (xmin * sin_coeff) + (ymax * cos_coeff);
    
    *outMinAreaRectBox.minAreaPointPtr(rectIdx, 0, 0) = bl_x;
    *outMinAreaRectBox.minAreaPointPtr(rectIdx, 0, 1) = bl_y;
    *outMinAreaRectBox.minAreaPointPtr(rectIdx, 1, 0) = tl_x;
    *outMinAreaRectBox.minAreaPointPtr(rectIdx, 1, 1) = tl_y;
    *outMinAreaRectBox.minAreaPointPtr(rectIdx, 2, 0) = tr_x;
    *outMinAreaRectBox.minAreaPointPtr(rectIdx, 2, 1) = tr_y;
    *outMinAreaRectBox.minAreaPointPtr(rectIdx, 3, 0) = br_x;
    *outMinAreaRectBox.minAreaPointPtr(rectIdx, 3, 1) = br_y;

}
void minAreaRectCUDA(ContourPtrCUDA<uint16_t>& inContourPointsData, MinAreaRectPtrCUDA<int>& outMinAreaRectBox, void* rotateCoeffs, void* numPointsInContourBuf, void* contoursToImages,  const int numContours, const int maxNumPointsInContour, const int numOfDegrees, const hipStream_t& stream)
{
    
    dim3 block(BLOCK * 8);
    dim3 grid(divUp(maxNumPointsInContour, block.x), numContours, numOfDegrees);
    // sm for rotate coeff
    size_t smem_size = 2 * numOfDegrees * sizeof(float);
    calculateRotateArea<<<grid, block, smem_size, stream>>>(inContourPointsData, outMinAreaRectBox, static_cast<float*>(rotateCoeffs), static_cast<int*>(numPointsInContourBuf), static_cast<int*>(contoursToImages));
    hipStreamSynchronize(stream);
    checkKernelErrors();
    dim3 grid1(numContours);
  
    findMinAreaAndAngle<<<grid1, block, smem_size, stream>>>(outMinAreaRectBox, numOfDegrees);
    hipStreamSynchronize(stream);
    checkKernelErrors();
}


__global__ void mergePatchKernel(ImagePtrCUDA<uchar> patchThreshold , ImagePtrCUDA<uchar> mergeThreshold, ImagePtrCUDA<float> patchRaw , ImagePtrCUDA<float> mergeRaw,  const ImgROI patchROI, const ImgROI mergeROI)
{
    int patch_x     = blockIdx.x * blockDim.x + threadIdx.x;
    int patch_y     = blockIdx.y * blockDim.y + threadIdx.y;

    if(patch_x < patchROI.w && patch_y < patchROI.h)
    {
        *mergeThreshold.ptr(0, mergeROI.y + patch_y, mergeROI.x + patch_x, 0) = *patchThreshold.ptr(0,patch_y+ patchROI.y,patch_x+patchROI.x, 0);
        *mergeRaw.ptr(0, mergeROI.y + patch_y, mergeROI.x + patch_x, 0) = *patchRaw.ptr(0,patch_y+ patchROI.y,patch_x+patchROI.x, 0);
    }
    return;

}


void patchMergeWarp(void* patchThresholdData, void* mergeThresholdData, void* patchOcdOutRawData, void* mergeOcdOutRawData, const nvinfer1::Dims& patchShape, const nvinfer1::Dims& mergeShape, const ImgROI& patchROI, const ImgROI& mergeROI, const hipStream_t& stream)
{   
    int patchBS = patchShape.d[0];
    int patchCh = patchShape.d[1];
    int patch_h = patchShape.d[2];
    int patch_w = patchShape.d[3];

    int mergeBS = mergeShape.d[0];
    int mergeCh = mergeShape.d[3];
    int merge_h = mergeShape.d[1];
    int merge_w = mergeShape.d[2];

    ImagePtrCUDA<uchar> patchThresholdPtr(patchBS, patch_h, patch_w, patchCh, static_cast<uchar*>(patchThresholdData), false);
    ImagePtrCUDA<float> patchRawPtr(patchBS, patch_h, patch_w, patchCh, static_cast<float*>(patchOcdOutRawData), false);
    ImagePtrCUDA<uchar> mergeThresholdPtr(mergeBS, merge_h, merge_w, mergeCh, static_cast<uchar*>(mergeThresholdData), true);
    ImagePtrCUDA<float> mergeRawPtr(mergeBS, merge_h, merge_w, mergeCh, static_cast<float*>(mergeOcdOutRawData), true);

    dim3 block(BLOCK,BLOCK);
    dim3 grid(divUp(patchROI.w, block.x),divUp(patchROI.h, block.y));

    mergePatchKernel<<<grid, block, 0 , stream>>>(patchThresholdPtr, mergeThresholdPtr, patchRawPtr, mergeRawPtr,  patchROI, mergeROI);
    hipStreamSynchronize(stream);
    checkKernelErrors();
}

//HWC as input
template<typename Dtype>
__device__ Dtype value_at_row_col_channel(Dtype *im, int rowIdx, int colIdx, int chIdx, int batchIdx, int colStride, int rowStride, int batchStride)
{
    Dtype *p = im + chIdx + ((int)colIdx) * colStride + ((int)rowIdx) * rowStride + ((int)batchIdx)*batchStride;
    return p[0];
}

template<typename Dtype>
__device__ void set_value_at_row_col_channel(Dtype vP_ch, Dtype *out_im, int rowIdx, int colIdx, int chIdx, int batchIdx, int colStride, int rowStride, int batchStride)
{
    Dtype *p = out_im + chIdx + ((int)colIdx) * colStride + ((int)rowIdx) * rowStride + ((int)batchIdx)*batchStride;
    p[0] = vP_ch;
}

template<typename Dtype>
__host__ __device__ __forceinline__  Dtype bordConstant(Dtype* src, int src_w, int src_h, int rowIdx, int colIdx, int chIdx, int batchIdx, int colStride, int rowStride, int batchStride)
{
    if((float)colIdx >= 0 && colIdx < src_w && (float)rowIdx >= 0 && rowIdx < src_h)
    {
        return value_at_row_col_channel(src, rowIdx, colIdx,  chIdx, batchIdx, colStride, rowStride, batchStride);
    }
    else 
    {
        return (Dtype)-1;
    }
}

template<typename Dtype>
__global__ void resize(
    Dtype *im, 
    int im_hh,
    int im_ww,
    int out_h,
    int out_w, 
    int channels, 
    int batchSize, 
    int dstColStride, 
    int dstRowStride, 
    int dstBatchStride, 
    float scale_w, 
    float scale_h, 
    Dtype *out_im
    )
{

    // float scale_w = float(im_ww)/float(out_w);
    // float scale_h = float(im_hh)/float(out_h);
    int srcColStride = channels;
    int srcRowStride = channels * im_ww;
    int srcBatchStride = channels * im_ww * im_hh;



    int x_out = blockIdx.x * blockDim.x + threadIdx.x;
    int y_out = blockIdx.y * blockDim.y + threadIdx.y;
    int c_out = threadIdx.z;
    int b_out = blockIdx.z;
    
    if (x_out < out_w && y_out < out_h)
    {

        float x_out_float = (float) x_out;
        float y_out_float = (float) y_out;
        float out = 0;
        
        //remapped to source image, called P
        float P_x_in_float = scale_w * (x_out_float+ 0.5) - 0.5;
        float P_y_in_float = scale_h * (y_out_float+ 0.5) - 0.5;
        
        float x1 = floorf(P_x_in_float);
        float y1 = floorf(P_y_in_float);
        
        float x2 = x1 + 1;
        float y2 = y1 + 1;
        
        Dtype src_reg = bordConstant(im, im_ww, im_hh, y1, x1, c_out, b_out, srcColStride, srcRowStride, srcBatchStride );
        out = out + src_reg * ((x2 - P_x_in_float) * (y2 - P_y_in_float));

        src_reg = bordConstant(im, im_ww, im_hh, y1, x2, c_out, b_out, srcColStride, srcRowStride, srcBatchStride );
        out = out + src_reg * ((P_x_in_float - x1) * (y2 - P_y_in_float));

        src_reg = bordConstant(im, im_ww, im_hh, y2, x1, c_out, b_out, srcColStride, srcRowStride, srcBatchStride );
        out = out + src_reg * ((x2 - P_x_in_float) * (P_y_in_float - y1));

        src_reg = bordConstant(im, im_ww, im_hh, y2, x2, c_out, b_out, srcColStride, srcRowStride, srcBatchStride );
        out = out + src_reg * ((P_x_in_float - x1) * (P_y_in_float - y1));

        
        set_value_at_row_col_channel((Dtype)out, out_im, y_out, x_out, c_out, b_out, dstColStride, dstRowStride, dstBatchStride);

    }
}

float
KeepAspectRatioResize(void* inData, void* outData, const nvinfer1::Dims& inShape,
                      const int32_t out_h, const int32_t out_w, const hipStream_t& stream)
{
    //Only support NHWC now
    int32_t batch_size = inShape.d[0];
    int32_t img_h = inShape.d[1];
    int32_t img_w = inShape.d[2];
    int32_t img_c = inShape.d[3];

    float ar_inp = float(img_w)/float(img_h);
    float ar_out = float(out_w)/float(out_h);

    int32_t new_width = 0;
    int32_t new_height = 0;
    float scale_w = 1.0f;
    float scale_h = 1.0f;
    float rescale = 1.0f;

    if (ar_inp >= ar_out)
    {
        new_width = out_w;
        new_height = new_width / ar_inp;
        rescale = float(img_w) / new_width;
    }
    else
    {
        new_height = out_h;
        new_width = new_height * ar_inp;
        rescale = float(img_h) / new_height;
    }

    scale_w = float(img_w) / float(new_width);
    scale_h = float(img_h) / float(new_height);

    int32_t dst_col_stride = img_c;
    int32_t dst_row_stride = img_c * out_w;
    int32_t dst_batch_stride = img_c * out_w * out_h;

    int32_t threads_num = 16;
    dim3 block(threads_num, threads_num, img_c);
    dim3 grid(divUp(new_width, threads_num), divUp(new_height, threads_num), batch_size);
    
    resize<uchar><<<grid, block, 0>>>(reinterpret_cast<uchar*>(inData), img_h, img_w, new_height, new_width, img_c,
                                               batch_size, dst_col_stride, dst_row_stride, dst_batch_stride,
                                               scale_w, scale_h, reinterpret_cast<uchar*>(outData));
    // hipStreamSynchronize(stream);
    // checkKernelErrors();
    return rescale;
}

__global__ void calculateBoxDistanceLauncher(float* boxCenters,float* boxLeftCenters ,float* boxRightCenters, short* letterMask, float* r2lDistance, const int letterNum)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    if(j < letterNum)
    {
        float box1_x =  boxCenters[2*i];
        float box1_y =  boxCenters[2*i + 1];
        float box2_x =  boxCenters[2*j];
        float box2_y =  boxCenters[2*j + 1];
        double slope = -(box2_y - box1_y) / (box2_x - box1_x + EXP);
        float radian = atan(slope);
        float angle = abs(radian/PI *180);

        // box1 left/right side center
        float box1_lx = boxLeftCenters[2*i];
        float box1_ly = boxLeftCenters[2*i + 1];
        float box1_rx = boxRightCenters[2*i];
        float box1_ry = boxRightCenters[2*i + 1];
        // box2 left/right side center
        float box2_lx = boxLeftCenters[2*j];
        float box2_ly = boxLeftCenters[2*j + 1];
        float box2_rx = boxRightCenters[2*j];
        float box2_ry = boxRightCenters[2*j + 1];

        float left2RightDis = sqrt( (box1_lx -box2_rx)*(box1_lx -box2_rx) + (box1_ly -box2_ry)*(box1_ly -box2_ry));
        float right2LeftDis = sqrt( (box1_rx -box2_lx)*(box1_rx -box2_lx) + (box1_ry -box2_ly)*(box1_ry -box2_ly));
        r2lDistance[i*MAX_LETTERS_IN_IMAGE + j] = right2LeftDis;
        if(angle < ANGLE_THRESHOLD )
        {
            letterMask[i*MAX_LETTERS_IN_IMAGE + j] = 1;
        }
        else
        {
            letterMask[i*MAX_LETTERS_IN_IMAGE + j] = -1;
        }
    }
    

}


void calculateBoxDistance(void* boxCenters, void* boxLeftCenters, void* boxRightCenters, void* letterMask, void* r2lDistance, const int letterNum, const hipStream_t& stream)
{

    dim3 block(BLOCK * divUp(letterNum, BLOCK));
    dim3 grid(letterNum);

    float* boxCentersMetric = static_cast<float*>(boxCenters);
    float* boxLeftCentersMetric = static_cast<float*>(boxLeftCenters);
    float* boxRightCentersMetric = static_cast<float*>(boxRightCenters);
    short* letterMaskMetric = static_cast<short*>(letterMask);
    float* r2lDistanceMetric = static_cast<float*>(r2lDistance);
    calculateBoxDistanceLauncher<<<grid, block,0, stream>>>(boxCentersMetric, boxLeftCentersMetric, boxRightCentersMetric, letterMaskMetric,r2lDistanceMetric, letterNum);
    // // sm for rotate coeff
    // size_t smem_size = 2 * numOfDegrees * sizeof(float);
    // calculateRotateArea<<<grid, block, smem_size, stream>>>(inContourPointsData, outMinAreaRectBox, static_cast<float*>(rotateCoeffs), static_cast<int*>(numPointsInContourBuf), static_cast<int*>(contoursToImages));
    // hipStreamSynchronize(stream);
    // checkKernelErrors();
    // dim3 grid1(numContours);
  
    // findMinAreaAndAngle<<<grid1, block, smem_size, stream>>>(outMinAreaRectBox, numOfDegrees);
    hipStreamSynchronize(stream);
    checkKernelErrors();
}

